#include "hip/hip_runtime.h"
#include "nn.hh"

#define CUDA_THREAD_INDEX (blockIdx.x * blockDim.x + threadIdx.x)
#define CUDA_THREAD_COUNT (gridDim.x * blockDim.x)

#define ASSERT_CUDA_ERROR() \
	do { \
		hipDeviceSynchronize(); \
		if (hipPeekAtLastError() != hipSuccess) \
		{ \
			std::cerr << __FILE__ << ":" << __LINE__ << ": " << hipGetErrorString(hipGetLastError()) << std::endl; \
			std::exit(EXIT_FAILURE); \
		} \
	} while (0)

//
// N_IN represents the size of the previous layer, while the number of threads
// corresponds to the size of the current layer.
// Ensure that N_IN * 4 is less than the available shared memory capacity.
// N_IN has to be equal to the number of threads per block.
//
template <std::uint32_t N_IN, float(*IN_ACTIVATION)(float)>
static __global__ void forward_fixed_layer(
	const float *__restrict__ biases,
	const float *__restrict__ weights,
	const float *__restrict__ in,
	const std::uint32_t in_stride,
	const std::uint32_t in_off,
	float *__restrict__ out,
	const std::uint32_t out_stride,
	const std::uint32_t out_off,
	const std::uint32_t batch_sz)
{
	__shared__ float shrd_in[N_IN];
	weights += (out_off + CUDA_THREAD_INDEX) * in_stride + in_off;
	in += in_off;
	out += out_off + CUDA_THREAD_INDEX;

	// TODO(petr): Loop through all neurons, instead of performing an out-of-bounds check.
	if (out_off + CUDA_THREAD_INDEX < out_stride) // TODO(petr): It should compare against n_out, instead of out_stride.
	{
		const float bias = biases[out_off + CUDA_THREAD_INDEX];
		for (std::uint32_t i = 0; i < batch_sz; i++)
		{
			float result = in_off ? *out : bias;

			__syncthreads();
			shrd_in[threadIdx.x] = IN_ACTIVATION(in[threadIdx.x]);
			__syncthreads();

			#pragma unroll
			for (std::uint32_t j = 0; j < N_IN; j++)
			{
				result += weights[j] * shrd_in[j];
			}

			*out = result;
			in += in_stride;
			out += out_stride;
		}
	}
}

template <std::uint32_t N_IN, float(*IN_ACTIVATION)(float)>
static const float *forward_layer(
	const float *__restrict__ biases,
	const float *__restrict__ weights,
	const float *__restrict__ in,
	const std::uint32_t n_in,
	float *__restrict__ out,
	const std::uint32_t n_out,
	const std::uint32_t batch_sz)
{
	std::uint32_t in_stride = (n_in + 127) & ~127;

	for (std::uint32_t out_off = 0; out_off < n_out; out_off += 40*N_IN)
	{
		for (std::uint32_t in_off = 0; in_off < n_in; in_off += N_IN)
		{
			forward_fixed_layer<N_IN, IN_ACTIVATION><<<40, N_IN>>>(
				biases,
				weights,
				in,
				in_stride,
				in_off,
				out,
				n_out,
				out_off,
				batch_sz);
		}
	}

	return out;
}

//
// N_OUT represents the size of the current layer, while the number of threads
// corresponds to the size of the previous layer.
// Ensure that N_OUT * 4 is less than the available shared memory capacity.
// N_OUT has to be equal to the number of threads per block.
//
template <std::uint32_t N_OUT, float(*IN_ACTIVATION_GRADIENT)(float)>
static __global__ void backward_fixed_layer(
	const float *__restrict__ weights,
	const float *__restrict__ gradients,
	const float *__restrict__ in,
	float *__restrict__ in_gradients,
	const std::uint32_t in_stride,
	const std::uint32_t in_off,
	const std::uint32_t out_stride,
	const std::uint32_t out_off,
	const std::uint32_t batch_sz)
{
	// TODO(petr): Lower the number of registers used by each thread.
	__shared__ float shrd_gradients[N_OUT];
	weights += out_off * in_stride + in_off + CUDA_THREAD_INDEX;
	gradients += out_off;
	in += in_off + CUDA_THREAD_INDEX;
	in_gradients += in_off + CUDA_THREAD_INDEX;

	if (in_off + CUDA_THREAD_INDEX < in_stride) // TODO(petr): It should compare against n_in, instead of in_stride.
	{
		for (std::uint32_t i = 0; i < batch_sz; i++)
		{
			float in_gradient = in_off ? *in_gradients : 0.0f;

			__syncthreads();
			shrd_gradients[threadIdx.x] = gradients[threadIdx.x];
			__syncthreads();

			#pragma unroll
			for (std::uint32_t j = 0; j < N_OUT; j++)
			{
				in_gradient += weights[j * in_stride] * shrd_gradients[j];
			}

			in_gradient *= IN_ACTIVATION_GRADIENT(*in);
			*in_gradients = in_gradient;

			in += in_stride;
			in_gradients += in_stride;
			gradients += out_stride;
		}
	}
}

template <std::uint32_t N_OUT, float(*IN_ACTIVATION_GRADIENT)(float)>
static void backward_layer(
	const float *__restrict__ weights,
	const float *__restrict__ gradients,
	const float *__restrict__ in,
	float *__restrict__ in_gradients,
	const std::uint32_t n_in,
	const std::uint32_t n_out,
	const std::uint32_t batch_sz)
{
	std::uint32_t in_stride = (n_in + 127) & ~127;
	std::uint32_t out_stride = (n_out + 127) & ~127;

	for (std::uint32_t in_off = 0; in_off < n_in; in_off += 40*N_OUT)
	{
		for (std::uint32_t out_off = 0; out_off < n_out; out_off += N_OUT)
		{
			backward_fixed_layer<N_OUT, IN_ACTIVATION_GRADIENT><<<40, N_OUT>>>(
				weights,
				gradients,
				in,
				in_gradients,
				in_stride,
				in_off,
				out_stride,
				out_off,
				batch_sz);
		}
	}
}

//
// N_IN represents the size of the previous layer, while the number of threads
// corresponds to the size of the current layer.
// Ensure that N_IN * 4 is less than the available shared memory capacity.
// N_IN has to be equal to the number of threads per block.
//
template <std::uint32_t N_IN, float(*IN_ACTIVATION)(float)>
static __global__ void update_fixed_layer(
	float *__restrict__ biases,
	float *__restrict__ weights,
	const float *__restrict__ gradients,
	const float *__restrict__ in,
	const std::uint32_t in_stride,
	const std::uint32_t in_off,
	const std::uint32_t out_stride,
	const std::uint32_t out_off,
	const float learning_rate,
	const std::uint32_t batch_sz)
{
	__shared__ float shrd_in[N_IN];
	weights += (out_off + CUDA_THREAD_INDEX) * in_stride + in_off;
	in += in_off;

	if (out_off + CUDA_THREAD_INDEX < out_stride)
	{
		float bias = biases[out_off + CUDA_THREAD_INDEX];
		for (std::uint32_t i = 0; i < batch_sz; i++)
		{
			float gradient = gradients[CUDA_THREAD_INDEX] * learning_rate;

			__syncthreads();
			shrd_in[threadIdx.x] = IN_ACTIVATION(in[threadIdx.x]);
			__syncthreads();

			#pragma unroll
			for (std::uint32_t j = 0; j < N_IN; j++)
			{
				weights[j] -= shrd_in[j] * gradient;
			}

			bias -= gradient;
			in += in_stride;
			gradients += out_stride;
		}
		biases[out_off + CUDA_THREAD_INDEX] = bias;
	}
}

template <std::uint32_t N_IN, float(*IN_ACTIVATION)(float)>
static void update_layer(
	float *__restrict__ biases,
	float *__restrict__ weights,
	const float *__restrict__ gradients,
	const float *__restrict__ in,
	const std::uint32_t n_in,
	const std::uint32_t n_out,
	const float learning_rate,
	const std::uint32_t batch_sz)
{
	std::uint32_t in_stride = (n_in + 127) & ~127;
	std::uint32_t out_stride = (n_in + 127) & ~127;

	for (std::uint32_t out_off = 0; out_off < n_out; out_off += 40*N_IN)
	{
		for (std::uint32_t in_off = 0; in_off < n_in; in_off += N_IN)
		{
			update_fixed_layer<N_IN, IN_ACTIVATION><<<40, N_IN>>>(
				biases,
				weights,
				gradients,
				in,
				in_stride,
				in_off,
				out_stride,
				out_off,
				learning_rate,
				batch_sz);
		}
	}
}

static __device__ __forceinline__ float _activation_stub(float x) { return x; }
static __device__ __forceinline__ float forward_tanh(float x)     { return tanhf(x); }
static __device__ __forceinline__ float backward_tanh(float x)    { float y = tanhf(x); return 1.0f - y*y; }
static __device__ __forceinline__ float forward_relu(float x)     { return x > 0.0f ? x : 0.0f; }
static __device__ __forceinline__ float backward_relu(float x)    { return x >= 0.0f ? 1.0f : 0.0f; }

Layer::Layer(std::uint32_t n_in, std::uint32_t n_out, std::uint32_t batch_sz, Activation in_activation)
	: n_in(n_in)
	, n_out(n_out)
	, batch_sz(batch_sz)
	, in_activation(in_activation)
{
	std::uint32_t in_stride = (n_in + 127) & ~127;
	std::uint32_t out_stride = (n_out + 127) & ~127;

	// TODO(petr): Randomize weights and biases.
	// TODO(petr): Are they correctly allocated?
	hipMalloc(&this->biases, n_out * sizeof(*this->biases));
	hipMalloc(&this->weights, n_out * in_stride * sizeof(*this->weights));
	hipMalloc(&this->out, n_out * this->batch_sz * sizeof(*this->out));
	hipMalloc(&this->gradients, out_stride * this->batch_sz * sizeof(*this->gradients));
}

Layer::~Layer()
{
	hipFree(&this->biases);
	hipFree(&this->weights);
	hipFree(&this->out);
	hipFree(&this->gradients);
}

const float *Layer::forward(const float *__restrict__ in) const
{
	switch (this->in_activation)
	{
	case Activation::Tanh:
		return forward_layer<128, forward_tanh>(this->biases, this->weights, in, this->n_in, this->out, this->n_out, this->batch_sz);
	case Activation::ReLU:
		return forward_layer<128, forward_relu>(this->biases, this->weights, in, this->n_in, this->out, this->n_out, this->batch_sz);
	default:
		return forward_layer<128, _activation_stub>(this->biases, this->weights, in, this->n_in, this->out, this->n_out, this->batch_sz);
	}
}

void Layer::backward(const float *__restrict__ in, float *__restrict__ in_gradients) const
{
	switch (this->in_activation)
	{
	case Activation::Tanh:
		return backward_layer<128, backward_tanh>(this->weights, this->gradients, in, in_gradients, this->n_in, this->n_out, this->batch_sz);
	case Activation::ReLU:
		return backward_layer<128, backward_relu>(this->weights, this->gradients, in, in_gradients, this->n_in, this->n_out, this->batch_sz);
	default:
		return backward_layer<128, _activation_stub>(this->weights, this->gradients, in, in_gradients, this->n_in, this->n_out, this->batch_sz);
	}
}

void Layer::update(const float *__restrict__ in, float learning_rate)
{
	switch (this->in_activation)
	{
	case Activation::Tanh:
		return update_layer<128, forward_tanh>(this->biases, this->weights, this->gradients, in, this->n_in, this->n_out, learning_rate, this->batch_sz);
	case Activation::ReLU:
		return update_layer<128, forward_relu>(this->biases, this->weights, this->gradients, in, this->n_in, this->n_out, learning_rate, this->batch_sz);
	default:
		return update_layer<128, _activation_stub>(this->biases, this->weights, this->gradients, in, this->n_in, this->n_out, learning_rate, this->batch_sz);
	}
}
