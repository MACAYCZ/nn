
#include <hip/hip_runtime.h>
#include <chrono>
#include <cstdint>
#include <iostream>

#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define CUDA_THREAD_INDEX (blockIdx.x * blockDim.x + threadIdx.x)
#define CUDA_THREAD_COUNT (gridDim.x * blockDim.x)

#define ASSERT_CUDA_ERROR() \
	do { \
		hipDeviceSynchronize(); \
		if (hipPeekAtLastError() != hipSuccess) \
		{ \
			std::cerr << __FILE__ << ":" << __LINE__ << ": " << hipGetErrorString(hipGetLastError()) << std::endl; \
			std::exit(EXIT_FAILURE); \
		} \
	} while (0)

constexpr std::uint32_t n_in = 128;
constexpr std::uint32_t n_out = 40*128;
constexpr std::uint32_t batch_sz = 4096;
constexpr std::uint32_t n_epochs = 100;
static_assert(n_out % 40 == 0);

//
// N_IN represents the size of the previous layer, while the number of threads
// corresponds to the size of the current layer.
// Ensure that N_IN * 4 is less than the available shared memory capacity.
//
template <std::uint32_t N_IN>
static __global__ void forward_fixed_layer(
	const float *__restrict__ _weights,
	const float *__restrict__ in,
	float *__restrict__ out,
	const std::uint32_t batch_sz)
{
	__shared__ float shrd_in[N_IN];
	const float *weights = _weights + CUDA_THREAD_INDEX * N_IN;
	for (std::uint32_t i = 0; i < batch_sz; i++)
	{
		// TODO(petr): It might be a good idea to make threadIdx.x equal to N_IN.
		for (std::uint32_t j = threadIdx.x; j < N_IN; j += blockDim.x)
		{
			shrd_in[j] = in[j];
		}
		__syncthreads();

		float result = 0.0f;
		#pragma unroll
		for (std::uint32_t j = 0; j < N_IN; j++)
		{
			result += weights[j] * shrd_in[j];
		}

		// TODO(petr): Remove the hardcoding of this activation function.
		result = tanhf(result);
		out[CUDA_THREAD_INDEX] += result;
		in += N_IN;
		out += CUDA_THREAD_COUNT;

		__syncthreads();
	}
}

int main(void)
{
	float *biases;
	float *weights;
	float *in;
	float *out;
	hipMalloc(&biases, n_out * sizeof(*biases));
	hipMalloc(&weights, n_out * n_in * sizeof(*weights));
	hipMalloc(&in, n_in * batch_sz * sizeof(*in));
	hipMalloc(&out, n_out * batch_sz * sizeof(*out));

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	for (std::size_t epoch = 0; epoch < n_epochs; epoch++)
	{
		hipMemcpyAsync(out, biases, n_out * sizeof(*biases), hipMemcpyDeviceToDevice);
		forward_fixed_layer<n_in><<<40, n_out / 40>>>(weights, in, out, batch_sz);
	}
	hipEventRecord(stop);
	ASSERT_CUDA_ERROR();

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	std::cout << "Elapsed time: " << milliseconds << "ms" << std::endl;

	hipFree(out);
	return 0;
}
