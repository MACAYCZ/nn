
#include <hip/hip_runtime.h>
#include <chrono>
#include <cstdint>
#include <iostream>

#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define CUDA_THREAD_INDEX (blockIdx.x * blockDim.x + threadIdx.x)
#define CUDA_THREAD_COUNT (gridDim.x * blockDim.x)

#define ASSERT_CUDA_ERROR() \
	do { \
		hipDeviceSynchronize(); \
		if (hipPeekAtLastError() != hipSuccess) \
		{ \
			std::cerr << __FILE__ << ":" << __LINE__ << ": " << hipGetErrorString(hipGetLastError()) << std::endl; \
			exit(EXIT_FAILURE); \
		} \
	} while (0)

constexpr std::uint32_t n_in = 64;
constexpr std::uint32_t n_out = 40*32;
constexpr std::uint32_t batch_sz = 4096;
constexpr std::uint32_t n_epochs = 10;

static_assert(n_out % 40 == 0);

template <std::uint32_t N_IN>
static __global__ void forward_fixed_layer(
	const float *_weights,
	const float *in,
	float *out,
	std::uint32_t batch_sz)
{
	const float *weights = _weights + CUDA_THREAD_INDEX * N_IN;
	for (std::uint32_t i = 0; i < batch_sz; i++)
	{
		float result = 0.0f;
		#pragma unroll
		for (std::uint32_t j = 0; j < N_IN; j++)
		{
			result += weights[j] * in[j];
		}
		// TODO(petr): Remove the hardcoding of this activation function.
		result = tanhf(result);
		out[CUDA_THREAD_INDEX] += result;
		in += N_IN;
		out += CUDA_THREAD_COUNT;
	}
}

int main(void)
{
	float *biases;
	float *weights;
	float *in;
	float *out;
	hipMalloc(&biases, n_out * sizeof(*biases));
	hipMalloc(&weights, n_out * n_in * sizeof(*weights));
	hipMalloc(&in, n_in * batch_sz * sizeof(*in));
	hipMalloc(&out, n_out * batch_sz * sizeof(*out));

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	for (std::size_t epoch = 0; epoch < n_epochs; epoch++)
	{
		hipMemcpyAsync(out, biases, n_out * sizeof(*biases), hipMemcpyDeviceToDevice);
		forward_fixed_layer<n_in><<<40, n_out / 40>>>(weights, in, out, batch_sz);
	}
	hipEventRecord(stop);
	ASSERT_CUDA_ERROR();

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	std::cout << "Elapsed time: " << milliseconds << "ms" << std::endl;

	hipFree(out);
	return 0;
}
